#include "hip/hip_runtime.h"
#include <ATen/ATen.h>


using namespace at;


namespace TensorUtils{

namespace {

template<typename scalar_t>
__global__ void fill_range_kernel(scalar_t* out, int64_t out_numel){
    auto index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < out_numel){
        out[index] = index;
    }
}
}

void fill_range_cuda_(Tensor t)
{
  // AT_ASSERT(t.type().is_cuda());

    const int threads_per_block = 256;
    const int blocks = t.numel()/threads_per_block + 1;

    auto scalar_type = t.type().scalarType();
    switch(scalar_type)
    {
        case ScalarType::Int: 
        fill_range_kernel<int32_t><<<blocks, threads_per_block>>>(t.data<int32_t>(), t.numel());
        break;

        case ScalarType::Long: 
        fill_range_kernel<int64_t><<<blocks, threads_per_block>>>(t.data<int64_t>(), t.numel());
        break;
        
        default:
        throw std::invalid_argument("Unrecognized Type!");
    }
}
    
} // namespace TensorUtils
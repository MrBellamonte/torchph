#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <limits>

#include "tensor_utils.cuh"
#include "param_checks_cuda.cuh"


using namespace at;

//TODO do proper namespacing and make header file

//TODO resolve template chaos should we always use long and never int? YES

//TODO change from by value args to by reference args 

//TODO remove corresponding cpp file and insert checked call wrappers in cu file

//TODO refactor python bindings in new file 

//TODO more assertions 
#pragma region find_merge_pairings


namespace CalcPersCuda{


namespace {

template <typename scalar_t>
__global__ void find_left_slicings_indices_cuda_kernel(
  scalar_t* __restrict__ input,
  scalar_t* __restrict__ output, 
  size_t input_size){ 

    const int index_middle = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_left   = index_middle - 1;
    const int index_right  = index_middle + 1;
    
    if (index_right < input_size){

      const int value_left = (index_left != -1) ? input[index_left] : -1; //OPTIMIZE: if we could
      // pad input with -1 on the left this conditional would be obsolete
      const int value_middle = input[index_middle];
      const int value_right = input[index_right];
      if (value_left != value_middle
          && 
          value_middle == value_right){
        output[index_middle] = index_middle;
      }
    }    
  }


template <typename scalar_t>
__global__ void find_right_slicings_indices_cuda_kernel(
  scalar_t* __restrict__ input,
  scalar_t* __restrict__ output, 
  size_t input_size){ 

    const int index_left   = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_middle = index_left + 1;
    const int index_right  = index_middle + 1;
    
    if (index_middle < input_size){

      const int value_left = input[index_left];
      const int value_middle = input[index_middle];
      const int value_right = (index_right < input_size) ? input[index_right] : (value_middle+1);//OPTIMIZE: if we could
      // pad input with inf on the right this conditional would be obsolete
      if (value_left == value_middle
          && 
          value_middle != value_right){
        output[index_middle] = index_middle + 1;
      }
    }    
  }

} // namespace


/**
 * @brief Finds the indices for slicing the sorted pivots values. 
 * Example:
 *    pivots.sort(0)[0] = [-1, -1, 2, 2, 2, 4, 4] -> [[2, 4], [5, 6]]
 * 
 * @tparam scalar_t 
 * @param pivots 
 * @return Tensor return.dtype() == scalar_t
 */
template <typename scalar_t>
Tensor find_slicing_indices_cuda_kernel_call(
    Tensor pivots) {
  Tensor output = zeros_like(pivots).fill_(-1);
  const int threads_per_block = 256;
  const int blocks = pivots.size(0)/threads_per_block + 1;

  find_left_slicings_indices_cuda_kernel<scalar_t><<<blocks, threads_per_block>>>(
    pivots.data<scalar_t>(), 
    output.data<scalar_t>(),
    pivots.size(0));

  find_right_slicings_indices_cuda_kernel<scalar_t><<<blocks, threads_per_block>>>(
    pivots.data<scalar_t>(), 
    output.data<scalar_t>(),
    pivots.size(0));

  output = output.masked_select(output.ge(0));
  output = output.view(IntList({output.size(0)/2, 2}));

  return output;
}  


namespace {

/**
 * @brief Implements a batch version of traditional slice for 
 * a input vector and a tensor which defines the slicings.
 * The output is a then of dimension
 * slicings.size(0) x (slicings[:, 1] - slicings[:, 0]).max()
 * 
 * @param p_input 
 * @param p_slicings 
 * @param p_return_value 
 * @param return_value_size_0 
 * @param return_value_size_1 
 */
__global__ void extract_slicings_cuda_kernel(
  int64_t* p_input,
  int32_t* p_slicings, 
  int64_t* p_return_value, 
  int64_t return_value_size_0, 
  int64_t return_value_size_1){
    
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < return_value_size_0){
      auto p_return_value_row = p_return_value + thread_id * return_value_size_1;
      const int slice_start = *(p_slicings + (thread_id * 2));
      const int slice_end = *(p_slicings + (thread_id * 2) + 1);

      for (int i = 0; i < slice_end - slice_start; i++){
        *(p_return_value_row + i) = *(p_input + slice_start + i);
      }
    }
}

/**
 * @brief Intended to be used on the output of 
 * extract_slicings_cuda_kernel. It reformats 
 * extraextracted_slices row-wise to merge-pairs 
 * format. E.g. 
 * row_i = [1, 2, 3] -> [[1,2], [1,3]]
 * 
 * @param extracted_slices 
 * @param extracted_slices_size_0 
 * @param extracted_slices_size_1 
 * @param lengths 
 * @param row_offset_for_thread 
 * @param return_value 
 */
__global__ void format_extracted_sorted_slicings_to_merge_pairs_kernel(
  int64_t* extracted_slices, 
  int64_t extracted_slices_size_0,
  int64_t extracted_slices_size_1,
  int32_t* lengths, 
  int64_t* row_offset_for_thread,
  int64_t* return_value){
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < extracted_slices_size_0){
      const int length = *(lengths + thread_id);
      const int row_offset = (thread_id > 0) ? *(row_offset_for_thread + thread_id - 1) : 0;
      auto extracted_slices_row = extracted_slices + thread_id*extracted_slices_size_1; 
      auto const first_col_value = *extracted_slices_row;

      auto return_value_row = return_value + 2*row_offset;
      for (int i = 0; i < length - 1; i++){
        *(return_value_row) = first_col_value;
        *(return_value_row + 1) = *(extracted_slices_row + i + 1);
        return_value_row = return_value_row + 2;
      }
    }
  }

} //namespace


Tensor sorted_pivot_indices_to_merge_pairs_cuda_kernel_call(Tensor input, Tensor slicings){
  // ASSERTION input.dtype() == int64
  // ASSERTION slicings.dtype() == int32
  // ASSERTION all(input.ge(0))
  // ASSERTION all(slicings.ge(0))
  // ASSERTION all(slicings[:, 0].leq(slicings[:, 1]))
  // ASSERTION slicings[:, 1].max() < input.size(0)

  auto lengths = (slicings.slice(1, 1, 2) - slicings.slice(1, 0, 1)).contiguous();
  auto max_lengths = Scalar(lengths.max()).to<int>(); 
  Tensor extracted_slicings = input.type().tensor({slicings.size(0), max_lengths});
  extracted_slicings.fill_(std::numeric_limits<int64_t>::max());

  const int threads_per_block_apply_slicings = 256;
  const int blocks_apply_slicings = slicings.size(0)/threads_per_block_apply_slicings + 1;
  extract_slicings_cuda_kernel<<<threads_per_block_apply_slicings, blocks_apply_slicings>>>(
    input.data<int64_t>(), 
    slicings.data<int32_t>(),
    extracted_slicings.data<int64_t>(),
    extracted_slicings.size(0),
    extracted_slicings.size(1)
  );

  auto extracted_slicings_sorted = std::get<0>(extracted_slicings.sort(1)).contiguous();

  auto lengths_minus_1 = lengths - lengths.type().scalarTensor(1);  
  auto row_offset_for_thread = lengths_minus_1.cumsum(0);

  auto merge_pairings_size_0 = Scalar(row_offset_for_thread[-1][0]).to<int>();
  auto merge_pairings = input.type().tensor({merge_pairings_size_0, 2});
  merge_pairings.fill_(-1);

  const int threads_per_block = 256;
  const int blocks = extracted_slicings_sorted.size(0)/threads_per_block + 1;

  format_extracted_sorted_slicings_to_merge_pairs_kernel<<<threads_per_block, blocks>>>(
      extracted_slicings_sorted.data<int64_t>(), 
      extracted_slicings_sorted.size(0),
      extracted_slicings_sorted.size(1), 
      lengths.data<int32_t>(), 
      row_offset_for_thread.data<int64_t>(),
      merge_pairings.data<int64_t>()
  );

  return merge_pairings;
  
}

class NoPairsException{
  public:
    NoPairsException() {}
    ~NoPairsException() {}
};


Tensor find_merge_pairings(
  Tensor pivots,
  int max_pairs = -1 ){

    CHECK_INPUT(pivots);
    assert(pivots.type().scalarType() == ScalarType::Int);

    if (max_pairs < 1){
      max_pairs = std::numeric_limits<int>::max();
    }
    auto sort_res = pivots.sort(0);
    auto sort_val = std::get<0>(sort_res);
    auto sort_ind = std::get<1>(sort_res);

    auto slicings = find_slicing_indices_cuda_kernel_call<int32_t>(sort_val).contiguous();

    Tensor merge_pairs; 
    if (slicings.size(0) != 0){         

      merge_pairs = sorted_pivot_indices_to_merge_pairs_cuda_kernel_call(sort_ind, slicings);
      // We sort the pairs such that pairs with smaller index come first.
      // This improves performance???
      if (merge_pairs.size(0) > max_pairs){

        sort_res = merge_pairs.slice(1, 0, 1).sort(0);
        sort_ind = std::get<1>(sort_res);
        sort_ind = sort_ind.slice(0, 0, max_pairs).squeeze();

        merge_pairs = merge_pairs.index_select(0, sort_ind);
        merge_pairs = merge_pairs.contiguous();
      }
    }
    else{
      throw NoPairsException();
    }

   return merge_pairs;
}


#pragma endregion 


#pragma region merge_columns


namespace{


  template <typename scalar_t>
  __device__ void merge_one_column_s(
    scalar_t* p_merger, 
    scalar_t* p_target, // the position of the target column, set to -1
    scalar_t* p_target_cache, // contains the copied values of target column 
    int boundary_array_size_1, 
    int* d_boundary_array_needs_resize 
  ){    
    // Assertion: comp_desc_sort_ba[:, -1] == -1 

    int p_target_increment_count = 0;

    while (true){
      if (*p_merger == -1 && *p_target_cache == -1){
        // both are -1, we have reached the end of meaningful entries -> break
        break;
      }

      if (*p_merger == *p_target_cache){
        // both values are the same but not -1 -> we eliminate 
        p_target_cache++;
        p_merger++;
      }
      else {

        if (*p_merger > *p_target_cache){
          //merger value is greater -> we take it 
          *p_target = *p_merger;
          p_merger++;
        }
        else
        {
          //target value is greate -> we take it 
          *p_target = *p_target_cache;
          p_target_cache++;
        }

        p_target++;  
        p_target_increment_count += 1;
      }          
    }

    if (p_target_increment_count > boundary_array_size_1/2){
      *d_boundary_array_needs_resize = 1; 
    }
  }


  template <typename scalar_t>
  __global__ void merge_columns_cuda_kernel(
      scalar_t* comp_desc_sort_ba,
      size_t descending_sorted_boundary_array_size_1, 
      scalar_t* cache, 
      int64_t* merge_pairings,
      size_t merge_pairings_size_0, 
      int* d_boundary_array_needs_resize
  ){
    //ASSERTION: cache.size(1) == comp_desc_sort_ba.size(1)
    const int thread_id = blockIdx.x*blockDim.x + threadIdx.x;   

    if (thread_id < merge_pairings_size_0){  

      const int filt_id_merger = merge_pairings[thread_id * 2];
      const int filt_id_target = merge_pairings[thread_id * 2 + 1];

      merge_one_column_s<int32_t>(
        comp_desc_sort_ba + (filt_id_merger * descending_sorted_boundary_array_size_1),
        comp_desc_sort_ba + (filt_id_target * descending_sorted_boundary_array_size_1),
        cache + (thread_id * descending_sorted_boundary_array_size_1), 
        descending_sorted_boundary_array_size_1,
        d_boundary_array_needs_resize
      );
    }
  }
  

} //namespace


template <typename scalar_t>
void merge_columns_cuda_kernel_call(
  Tensor comp_desc_sort_ba,
  Tensor merge_pairings, 
  int* h_boundary_array_needs_resize
)
{
  const int threads_per_block = 32;
  const int blocks = merge_pairings.size(0)/threads_per_block + 1;

  auto targets = merge_pairings.slice(1, 1).squeeze();  
  
  // fill cache for merging ... 
  //  TODO optimize: we do not need all columns it is enough to take des...array.size(1)/2 + 1 
  //  ATTENTION if we do this we have to inform merge_columns_cuda_kernel about this!!!
  auto cache = comp_desc_sort_ba.index_select(0, targets);
  
  auto size = sizeof(int);
  int* d_boundary_array_needs_resize;
  hipMalloc(&d_boundary_array_needs_resize, size);
  hipMemcpy(d_boundary_array_needs_resize, h_boundary_array_needs_resize, size, hipMemcpyHostToDevice);

  // reset content of target columns 
  comp_desc_sort_ba.index_fill_(0, targets, -1);

  merge_columns_cuda_kernel<int32_t><<<blocks, threads_per_block>>>(
    comp_desc_sort_ba.data<int32_t>(), 
    comp_desc_sort_ba.size(1), 
    cache.data<int32_t>(),
    merge_pairings.data<int64_t>(), 
    merge_pairings.size(0), 
    d_boundary_array_needs_resize
  );

  hipDeviceSynchronize();
  hipMemcpy(h_boundary_array_needs_resize, d_boundary_array_needs_resize, size, hipMemcpyDeviceToHost);

  hipFree(d_boundary_array_needs_resize);
}


Tensor resize_boundary_array(
  Tensor comp_desc_sort_ba){
    auto tmp = empty_like(comp_desc_sort_ba);
    tmp.fill_(-1);
    auto new_ba = cat(TensorList({comp_desc_sort_ba, tmp}), 1);
    return new_ba.contiguous();
}


Tensor merge_columns(
  Tensor comp_desc_sort_ba, 
  Tensor merge_pairings){   

    CHECK_INPUT(comp_desc_sort_ba);
    assert(comp_desc_sort_ba.type().scalarType() == ScalarType::Int);
    CHECK_INPUT(merge_pairings);
    assert(merge_pairings.type().scalarType() == ScalarType::Long);
    
    int boundary_array_needs_resize = 0;
    int* h_boundary_array_needs_resize = &boundary_array_needs_resize;    

    merge_columns_cuda_kernel_call<int32_t>(
      comp_desc_sort_ba,
      merge_pairings, 
      h_boundary_array_needs_resize
    );
  
    if (*h_boundary_array_needs_resize == 1){
      comp_desc_sort_ba = resize_boundary_array(comp_desc_sort_ba);
    }
    
    return comp_desc_sort_ba;
  }


#pragma endregion


#pragma region read_barcodes


std::vector<std::vector<Tensor> > read_barcodes(
  Tensor pivots, 
  Tensor simplex_dimension, 
  int max_dimension){

    CHECK_INPUT(pivots);
    assert(pivots.type().scalarType() == ScalarType::Int);
    CHECK_INPUT(simplex_dimension);
    assert(simplex_dimension.type().scalarType() == ScalarType::Int);
    std::vector<Tensor> ret_non_ess; 
    std::vector<Tensor> ret_ess;
    simplex_dimension = simplex_dimension.unsqueeze(1);    

    auto range = empty_like(pivots);
    TensorUtils::fill_range_cuda_(range); 

    auto pool_for_barcodes_non_essential = cat({pivots, range}, 1);
    auto mask_pivot = pivots.ge(0);
    
    // all dimenions mask non essential ... 
    auto mask_non_essential = mask_pivot.expand({-1, 2});

    // all dimensions mask essential ...
    auto mask_no_pivot = pivots.le(-1); 
    auto mask_rows_with_no_lowest_one = ones_like(mask_no_pivot);
    auto row_indices_with_lowest_one = pivots.masked_select(mask_pivot).toType(ScalarType::Long);

    mask_rows_with_no_lowest_one.index_fill_(0, row_indices_with_lowest_one, 0);

    auto mask_ess = mask_no_pivot.__and__(mask_rows_with_no_lowest_one);

    for (int dim=0; dim <= max_dimension; dim++){
      
      // non essentials ...
      auto mask_dim = simplex_dimension.eq(dim + 1);
      auto mask_non_essential_dim = mask_non_essential.__and__(mask_dim.expand({-1, 2}));
      auto barcodes_non_essential_dim = pool_for_barcodes_non_essential.masked_select(mask_non_essential_dim).view({-1, 2});
      
      ret_non_ess.push_back(barcodes_non_essential_dim);

      // essentials ...
      auto mask_dim_ess = simplex_dimension.eq(dim);
      auto mask_essential_dim = mask_ess.__and__(mask_dim_ess); 
      auto barcode_birth_times_essential_dim = range.masked_select(mask_essential_dim).view({-1, 1});

      ret_ess.push_back(barcode_birth_times_essential_dim);
    } 

    return std::vector<std::vector<Tensor> >({ret_non_ess, ret_ess});
  }


#pragma endregion 


std::vector<std::vector<Tensor> > calculate_persistence(   
    Tensor comp_desc_sort_ba, 
    Tensor ind_not_reduced, //TODO rename parameter accordingly to python binding 
    Tensor simplex_dimension,
    int max_dimension,
    int max_pairs = -1
    ) {

  CHECK_INPUT(comp_desc_sort_ba);
  assert(comp_desc_sort_ba.type().scalarType() == ScalarType::Int);
  CHECK_INPUT(ind_not_reduced);
  assert(ind_not_reduced.type().scalarType() == ScalarType::Long);
  CHECK_INPUT(simplex_dimension);
  assert(simplex_dimension.type().scalarType() == ScalarType::Int);

  assert(comp_desc_sort_ba.size(0) == ind_not_reduced.size(0));
  assert(ind_not_reduced.ndimension() == 1);
  assert(simplex_dimension.ndimension() == 1);
  
  int iterations = 0;

  // auto ind_not_reduced = comp_desc_sort_ba.type()
  //   .toScalarType(ScalarType::Long).tensor({simplex_dimension.size(0), 1});
  // fill_range_cuda_(ind_not_reduced);
  
  // auto tmp_pivots = comp_desc_sort_ba.slice(1, 0, 1).contiguous();
  auto scalar_0 = comp_desc_sort_ba.type().scalarTensor(0);

  // Tensor mask_not_reduced = tmp_pivots.ge(scalar_0);

  // ind_not_reduced = ind_not_reduced.masked_select(mask_not_reduced).contiguous();

  // comp_desc_sort_ba =
  //   comp_desc_sort_ba.index_select(0, ind_not_reduced).contiguous();

  Tensor mask_not_reduced, pivots, merge_pairings, new_ind_not_reduced;
  while(true){

    pivots = comp_desc_sort_ba.slice(1, 0, 1).contiguous();

    try{

      merge_pairings = find_merge_pairings(pivots, max_pairs);   

    }
    catch(NoPairsException& e){

      std::cout << "Reached end of reduction after " << iterations << " iterations" << std::endl;
      break;

    }

    comp_desc_sort_ba = merge_columns(comp_desc_sort_ba, merge_pairings);

    new_ind_not_reduced = comp_desc_sort_ba.type()
      .toScalarType(ScalarType::Long).tensor({comp_desc_sort_ba.size(0), 1});
    TensorUtils::fill_range_cuda_(new_ind_not_reduced);
    
    pivots = comp_desc_sort_ba.slice(1, 0, 1).contiguous();
    mask_not_reduced = pivots.ge(scalar_0);
    new_ind_not_reduced = new_ind_not_reduced.masked_select(mask_not_reduced).contiguous();

    comp_desc_sort_ba =
      comp_desc_sort_ba.index_select(0, new_ind_not_reduced).contiguous();
    // pivots = pivots.index_select(0, new_ind_not_reduced).contiguous();

    ind_not_reduced = ind_not_reduced.index_select(0, new_ind_not_reduced);

    iterations++;

  }

  auto real_pivots = pivots.type().tensor({simplex_dimension.size(0), 1}).fill_(
    pivots.type().scalarTensor(-1)
  );
  real_pivots.index_copy_(0, ind_not_reduced, pivots); 

  auto barcodes = read_barcodes(real_pivots, simplex_dimension, max_dimension);
  return barcodes;
}


Tensor my_test_f(Tensor t){
  auto ret = zeros_like(t);

  // my_test_kernel<<<1, 32>>>(t);

  return ret;
}


} // namespace CalcPersCuda
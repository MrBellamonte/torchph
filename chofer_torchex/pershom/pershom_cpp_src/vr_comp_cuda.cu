#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <limits>

#include "param_checks_cuda.cuh"
#include "tensor_utils.cuh"
#include "calc_pers_cuda.cuh"

using namespace at;


//TODO extract in other file 
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}


namespace VRCompCuda {


#pragma region binomal_table

__device__ int64_t binom_coeff(int64_t n, int64_t k){
    int64_t res = 1; 

    if ( k > n) return 0; 
 
    // Since C(n, k) = C(n, n-k)
    if ( k > n - k ){
        k = n - k;
    }
 
    // Calculate value of [n * (n-1) *---* (n-k+1)] / [k * (k-1) *----* 1]
    for (int64_t i = 0; i < k; i++)
    {
        res *= (n-i);
        res /= (i + 1);
    }
 
    return res;
}


__global__ void binomial_table_kernel(int64_t* out, int64_t max_n, int64_t max_k){
    int64_t r = blockIdx.x*blockDim.x + threadIdx.x;
    int64_t c = blockIdx.y*blockDim.y + threadIdx.y;

    if (r < max_k && c < max_n){

        out[r*max_n + c] = binom_coeff(c, r + 1);

    }
}


/**
 * @brief 
 * 
 * @param max_n 
 * @param max_k 
 * @param type 
 * @return Tensor [max_k, max_n] where return[i, j] = binom(j, i+1)
 */
Tensor binomial_table(int64_t max_n, int64_t max_k, const Type& type){
 
    
    auto ret = type.toScalarType(ScalarType::Long).tensor({max_k, max_n}); //LBL: creation 

    dim3 threads_per_block = dim3(8, 8);
    dim3 num_blocks= dim3(max_k/threads_per_block.y + 1, max_n/threads_per_block.x + 1);    


    binomial_table_kernel<<<num_blocks, threads_per_block>>>(
        ret.data<int64_t>(),
        max_n, 
        max_k);      

    cudaCheckError();   

    return ret; 
}

#pragma endregion 

#pragma region write combinations table to tensor

int64_t binom_coeff_cpu(int64_t n, int64_t k){
    int64_t res = 1; 

    if ( k > n) return 0; 
 
    // Since C(n, k) = C(n, n-k)
    if ( k > n - k ){
        k = n - k;
    }
 
    // Calculate value of [n * (n-1) *---* (n-k+1)] / [k * (k-1) *----* 1]
    for (int64_t i = 0; i < k; i++)
    {
        res *= (n-i);
        res /= (i + 1);
    }
 
    return res;
}

// Unranking of a comination, c.f., https://en.wikipedia.org/wiki/Combinatorial_number_system
__device__ void unrank_combination(
    int64_t* out, // writing to this, assert length(out) == r
    const int64_t N, // rank of the combination 
    const int64_t max_n, // cobinations of elements < max_n
    const int64_t r, // number of combinations
    int64_t* const binom_table //cache of combinations, assert binom_table[i,j] == binom(j,i)
    ){

    int64_t* bt_row; 
    int64_t rest = N; 
    bool broken = false; 

    for (int64_t i=0; i<r;i++){
        bt_row = &binom_table[(r - i - 1)*max_n];

        for (int64_t j=0; j < max_n - 1; j++){
            if (bt_row[j] <= rest && bt_row[j+1] > rest){
                rest = rest - bt_row[j];
                out[i] = j;  
                broken = true;
                break; 
            }
        }

        if (!broken) {
            out[i] = max_n - 1; 
            rest = rest - bt_row[max_n -1];
        }
        
    }
}


// writes the next combination into out, e.g., out = [3, 2, 1] -> out = [4, 2, 1] 
__device__ void next_combination(int64_t* out, int64_t r){

    // If we have to increase not the first digit ... 
    for (int64_t i = 0; i < r; i++){        
        if (out[r - i - 2] > out[r - i - 1] + 1){
            out[r - i - 1] += 1;
           
            // fill the following digits with the smallest ordered sequence ... 
            for (int64_t j=0; j < i; j++){
                out[r - j - 1] = j;
            }
            return;
        }
    }

    // If the first digit has to be increased ...
    out[0] += 1;    
    // fill the following digits with the smallest ordered sequence ... 
    for (int64_t j=0; j < r - 1; j++){
            out[r - j - 1] = j;
    }
  
}


__global__ void write_combinations_to_tensor_kernel(
    int64_t* out, 
    const int64_t out_row_offset, 
    const int64_t out_row_stride, 
    const int64_t additive_constant, // additive constant which is added to the digits of each combination
    const int64_t max_n, 
    const int64_t r, 
    int64_t* binom_table, 
    const int64_t n_comb_by_thread, 
    const int64_t n_max_over_r){

    int64_t thread_id = blockIdx.x*blockDim.x + threadIdx.x;


    if (thread_id*n_comb_by_thread < binom_coeff(max_n, r)){ // TODO use parameter instead of binom_coeff call
        int64_t* comb = new int64_t[r]; 
        unrank_combination(comb, thread_id*n_comb_by_thread, max_n, r, binom_table);

        for (int64_t i = 0; i < n_comb_by_thread; i++){

            if (thread_id*n_comb_by_thread + i >= n_max_over_r) break; 

            for (int64_t j = 0; j < r; j++){
                out[out_row_stride * (out_row_offset + thread_id*n_comb_by_thread + i ) + j] 
                    = comb[j] + additive_constant;
            }     

            next_combination(comb, r);
        }

        delete[] comb;
    }

    __syncthreads();

}

/*
Writes all combinations of {0, ... , max_n -1} of length r 
to out in lexicographical order. 
Example max_n = 4, r = 3, off_set = 1, additive_constant = 0

     out     ->      out 

-1 -1 -1 -1     -1 -1 -1 -1
-1 -1 -1 -1      2  1  0 -1 
-1 -1 -1 -1      3  1  0 -1
-1 -1 -1 -1      3  2  0 -1
-1 -1 -1 -1      3  2  1 -1
-1 -1 -1 -1     -1 -1 -1 -1
*/ 
void write_combinations_table_to_tensor(
    const Tensor& out, 
    const int64_t out_row_offset, // the 
    const int64_t additive_constant, // constant added to each digit of the combination
    const int64_t max_n, 
    const int64_t r
    ){

    CHECK_SMALLER_EQ(r, max_n); 
    const int64_t n_max_over_r = binom_coeff_cpu(max_n, r);

    CHECK_SMALLER_EQ(0, out_row_offset);
    CHECK_SMALLER_EQ(n_max_over_r + out_row_offset, out.size(0));
    CHECK_SMALLER_EQ(r, out.size(1));  
    CHECK_EQUAL(out.ndimension(), 2);


    const auto bt = binomial_table(max_n, r, out.type());
    const int n_comb_by_thread = 100; //TODO optimize
    int threads_per_block = 64; //TODO optimize

    int blocks = n_max_over_r/(threads_per_block*n_comb_by_thread) + 1;

    write_combinations_to_tensor_kernel<<<blocks, threads_per_block>>>(
        out.data<int64_t>(), 
        out_row_offset, 
        out.size(1), 
        additive_constant, 
        max_n, 
        r, 
        bt.data<int64_t>(),
        n_comb_by_thread, 
        n_max_over_r
    );

    cudaCheckError(); 
}

#pragma endregion 


Tensor l1_norm_distance_matrix(const Tensor& points){
    Tensor ret = points.unsqueeze(1).expand({points.size(0), points.size(0), points.size(1)});

    return (ret.transpose(0, 1) - ret).abs().sum(2); 
}


std::tuple<Tensor, Tensor> get_boundary_and_filtration_info_dim_1(
    const Tensor & point_cloud, 
    double max_ball_radius){

    Tensor ba_dim_1, filt_val_vec_dim_1; 
    auto n_edges = binom_coeff_cpu(point_cloud.size(0), 2); 
    ba_dim_1 = point_cloud.type().toScalarType(ScalarType::Long).tensor({n_edges, 2}); 

    write_combinations_table_to_tensor(ba_dim_1, 0, 0, point_cloud.size(0)/*=max_n*/, 2/*=r*/);

    auto distance_matrix = l1_norm_distance_matrix(point_cloud); 

    hipStreamSynchronize(0); // ensure that write_combinations_table_to_tensor call has finished
    // building the vector containing the filtraiton values of the edges 
    // in the same order as they appear in ba_dim_1...
    auto x_indices = ba_dim_1.slice(1, 0, 1).squeeze(); 
    auto y_indices = ba_dim_1.slice(1, 1, 2); 

    // filling filtration vector with edge filtration values ... 
    filt_val_vec_dim_1 = distance_matrix.index_select(0, x_indices);
    filt_val_vec_dim_1 = filt_val_vec_dim_1.gather(1, y_indices);
    filt_val_vec_dim_1 = filt_val_vec_dim_1.squeeze(); // 

    // reduce to edges with filtration value <= max_ball_radius...
    if (max_ball_radius > 0){
        auto i_select = filt_val_vec_dim_1.le(point_cloud.type().scalarTensor(max_ball_radius)).nonzero().squeeze(); 
        if (i_select.numel() ==  0){
            ba_dim_1 = ba_dim_1.type().tensor({0});
            filt_val_vec_dim_1 = filt_val_vec_dim_1.type().tensor({0}); 
        }
        else{
            ba_dim_1 = ba_dim_1.index_select(0, i_select);
            filt_val_vec_dim_1 = filt_val_vec_dim_1.index_select(0, i_select); 
        }
    }

    return std::make_tuple(ba_dim_1, filt_val_vec_dim_1);
}


std::tuple<Tensor, Tensor> get_boundary_and_filtration_info(
    const Tensor & filt_vals_prev_dim, 
    int64_t dim){

    auto n_dim_min_one_simplices = filt_vals_prev_dim.size(0); 

    Tensor new_boundary_info, new_filt_vals;

    if (n_dim_min_one_simplices < dim + 1){
        // There are not enough dim-1 simplices ...
        new_boundary_info = filt_vals_prev_dim.type().toScalarType(ScalarType::Long).tensor({0, dim + 1});
        new_filt_vals = filt_vals_prev_dim.type().tensor({0});
    }
    else{
        // There are enough dim-1 simplices ...
        auto n_new_simplices = binom_coeff_cpu(n_dim_min_one_simplices, dim + 1); 
        auto n_simplices_prev_dim = filt_vals_prev_dim.size(0); 

        new_boundary_info = filt_vals_prev_dim.type().toScalarType(ScalarType::Long).tensor({n_new_simplices, dim + 1}); 

        // write combinations ... 
        write_combinations_table_to_tensor(new_boundary_info, 0, 0, n_simplices_prev_dim, dim + 1); 
        hipStreamSynchronize(0); 

        auto bi_cloned = new_boundary_info.clone(); // we have to clone here other wise auto-grad does not work!
        new_filt_vals = filt_vals_prev_dim.expand({n_new_simplices, filt_vals_prev_dim.size(0)});
        new_filt_vals = new_filt_vals.gather(1, bi_cloned); 
        new_filt_vals = std::get<0>(new_filt_vals.max(1));

        // If we have just one simplex of the current dimension this
        // condition avoids that new_filt_vals is squeezed to a 0-dim 
        // Tensor
        if (new_filt_vals.ndimension() != 1){      
            new_filt_vals = new_filt_vals.squeeze(); 
        }
    }

    return std::make_tuple(new_boundary_info, new_filt_vals); 
}


//TODO refactor 
std::vector<Tensor> vr_l1_generate_calculate_persistence_args(
    const Tensor& point_cloud,
    int64_t max_dimension, 
    double max_ball_radius
    ){

    CHECK_TENSOR_CUDA_CONTIGUOUS(point_cloud);
    CHECK_SMALLER_EQ(max_dimension + 1, point_cloud.size(0)); 
    CHECK_SMALLER_EQ(0, max_ball_radius);


    std::vector<Tensor> ret;
    Type& Long = point_cloud.type().toScalarType(ScalarType::Long);

    // 1. generate boundaries and filtration values ...

    // boundary_and_filtration_info_by_dim[i] == (enumerated boundary combinations, filtration values) of 
    // dimension i + 1. 
    std::vector<std::tuple<Tensor, Tensor>> boundary_and_filtration_by_dim;

    boundary_and_filtration_by_dim.push_back(
        get_boundary_and_filtration_info_dim_1(point_cloud, max_ball_radius)
    );

    for (int dim = 2; dim <= max_dimension; dim++){
        auto filt_vals_prev_dim = std::get<1>(boundary_and_filtration_by_dim.at(dim - 1 - 1));

        boundary_and_filtration_by_dim.push_back(
            get_boundary_and_filtration_info(filt_vals_prev_dim, dim)
        );
    }

    // 2. Create helper structure which contains meta info about simplex numbers ... 
    int64_t n_non_vertex_simplices = 0;
    int64_t n_simplices = point_cloud.size(0); 
    std::vector<int64_t> n_simplices_by_dim; 
    n_simplices_by_dim.push_back(point_cloud.size(0)); 

    for (int i = 0; i < boundary_and_filtration_by_dim.size(); i++){
        auto boundary_info = std::get<0>(boundary_and_filtration_by_dim.at(i));
        n_non_vertex_simplices += boundary_info.size(0); 
        n_simplices += boundary_info.size(0); 
        n_simplices_by_dim.push_back(boundary_info.size(0)); 
    }

    // TODO returning in mid of function is not nice. Can we improve this? 
    // If there are only vertices, we return the empty vector 
    // and let the caller handle the problem ... 
    if (n_non_vertex_simplices == 0){
        return ret; 
    }

    // 3. Make simplex id's compatible within dimensions ... 
    /*    
    In order to keep indices in the boundary info tensors 
    compatible within dimensions we have to add an offset
    to the enumerated combinations, starting with 
    dimension 2 simplices (the boundaries of dim 1 simplices are vertices, 
    hence the enumeration of the boundary combinations is valid)
    */
    auto index_offset = n_simplices_by_dim.at(0);
    for (int i=1; i < boundary_and_filtration_by_dim.size(); i++){
        auto boundary_info = std::get<0>(boundary_and_filtration_by_dim.at(i)); 
        boundary_info.add_(index_offset); 

        auto n_simplices_in_prev_dim = std::get<0>(boundary_and_filtration_by_dim.at(i-1)).size(0); 
        index_offset += n_simplices_in_prev_dim;
    }    


    // 4. Create simplex_dimension vector ... 
    auto simplex_dimension = Long.tensor(n_non_vertex_simplices + n_simplices_by_dim.at(0)); 

    {
        int64_t copy_offset = 0; 
        for (int i = 0; i <= max_dimension; i++){
            simplex_dimension.slice(0, copy_offset, copy_offset + n_simplices_by_dim.at(i)).fill_(i); 
            copy_offset += n_simplices_by_dim.at(i); 
        }
    }


    // 5. Create filtration vector ... 
    Tensor filtration_values_vector;
    {
        std::vector<Tensor> filt_values_non_vertex_simplices; 
        for (int i = 0; i < boundary_and_filtration_by_dim.size(); i++){
        
            auto filt_vals = std::get<1>(boundary_and_filtration_by_dim.at(i));
            filt_values_non_vertex_simplices.push_back(filt_vals);  
        } 

        filtration_values_vector = cat(filt_values_non_vertex_simplices, 0); 
    }    

    // This is a dirty hack to ensure that simplices do not occour before their boundaries 
    // in the filtration. As the filtration is raised to higher dimensional simplices by 
    // taking the maxium of the involved edge filtration values and sorting does not guarantee
    // a specific ordering in case of equal values we are forced to ensure a well defined 
    // filtration by adding an increasing epsilon to each dimension. Later this has to be 
    // substracted again. 
    // Example: f([1,2,3]) = max(f([1,2]), f([3,1]), f([2,3])) --> w.l.o.g. f([1,2,3]) == f([1,2])
    // Hence we set f([1,2,3]) = f([1,2]) + epsilon
    auto filt_add_hack_values = filtration_values_vector.type().tensor({filtration_values_vector.size(0)}).fill_(0);
    
    {
        if (max_dimension >= 2 && n_simplices_by_dim.at(2) > 0){
            
            // we take epsilon of float to ensure that it is well defined even if 
            // we decide to alter the floating point type of the filtration values 
            // realm 
            float add_const_base_value = 100 * std::numeric_limits<float>::epsilon(); // multily with 100 to be save against rounding issues
            auto copy_offset = n_simplices_by_dim.at(1); 

            for (int dim = 2; dim <= max_dimension; dim++){
                filt_add_hack_values.slice(0, copy_offset, copy_offset + n_simplices_by_dim.at(dim))
                    .fill_(add_const_base_value); 

                add_const_base_value += add_const_base_value; 
                copy_offset += n_simplices_by_dim.at(dim); 
            }

            filtration_values_vector += filt_add_hack_values;
        }

        filt_add_hack_values = filt_add_hack_values.clone();
    
    }

    //6 Do sorting ...
    
    auto sort_filt_res = filtration_values_vector.sort(0);
    auto sorted_filtration_values_vector = std::get<0>(sort_filt_res);
    auto sort_i_filt = std::get<1>(sort_filt_res); 

    // revert filtration hack if necessary ...
    if (max_dimension >= 2 && n_simplices_by_dim.at(2) > 0){
        filt_add_hack_values = filt_add_hack_values.index_select(0, sort_i_filt); 
        sorted_filtration_values_vector -= filt_add_hack_values;
    }
    // now the filtration is cleaned and we can continue. 

    // Simplex ids in boundary_array entries include vertices.
    // As filtration_value_vector so far starts with edges we have to take care of this. 
    auto dim_0_filt_values = sorted_filtration_values_vector.type().zeros({n_simplices_by_dim.at(0)}); 
    sorted_filtration_values_vector = cat({dim_0_filt_values, sorted_filtration_values_vector}, 0); 
  

    // Sort simplex_dimension ...
    simplex_dimension.slice(0, n_simplices_by_dim.at(0)) = 
        simplex_dimension.slice(0, n_simplices_by_dim.at(0)).index_select(0, sort_i_filt);

    // Copy boundary_info of each dimension into the final boundary array ... 
    auto boundary_array = point_cloud.type().toScalarType(ScalarType::Long)
        .tensor({n_non_vertex_simplices, 2*(max_dimension + 1)});

    {
        boundary_array.fill_(-1); 

        // copy edges ... 
        auto edge_boundary_info = std::get<0>(boundary_and_filtration_by_dim.at(0));
        boundary_array.slice(0, 0, n_simplices_by_dim.at(1)).slice(1, 0, 2) = edge_boundary_info; 

        // copy higher dimensional simplices
        if (max_dimension >= 2){
            // we need a look up table which lets us change the simplex ids we get from the initial 
            // enumeration (write_combinations_table_to_tensor) to the id the have w.r.t. the ordering
            // of the filtration values. We create this table now ...

            // This gives us the mapping id -> new_id w.r.t. sorting by filtration values ...
            auto look_up_table_row = std::get<1>(sort_i_filt.sort(0));

            // look_up_table_row is yet based on id's without vertices, we adapt theis now ...
            auto dummy_sort_indices = sort_i_filt.type().tensor({n_simplices_by_dim.at(0)}).fill_(std::numeric_limits<int64_t>::max());
            look_up_table_row = look_up_table_row + n_simplices_by_dim.at(0); 

            // as vertices have no boundary we will never select a value of the first #vertices entries, 
            // but we need look_up_table_row.size(0) == #simplices in order to get a consistent mapping...
            look_up_table_row = cat({dummy_sort_indices, look_up_table_row}, 0); 

            int64_t copy_offset = n_simplices_by_dim.at(1);             

            for (int i = 1; i < max_dimension; i++){

                auto boundary_info = std::get<0>(boundary_and_filtration_by_dim.at(i)); 

                if (boundary_info.size(0) == 0){
                    continue; 
                }

                auto look_up_table = look_up_table_row.expand({boundary_info.size(0), look_up_table_row.size(0)});  

                // Apply ordering to row content ... 
                boundary_info = look_up_table.gather(1, boundary_info); 

                // Apply ordering to rows ...
                boundary_info = std::get<0>(boundary_info.sort(1, /*descending=*/true));

                boundary_array.slice(0, copy_offset, copy_offset + boundary_info.size(0)).slice(1, 0, boundary_info.size(1))
                     = boundary_info; 

                copy_offset += boundary_info.size(0); 
            }
        }
    }

    // Sort boundary_array rows ...
    boundary_array = boundary_array.index_select(0, sort_i_filt);  

    //7. generate ba_row_i_to_bm_col_i
    auto ba_row_i_to_bm_col_i = boundary_array.type().tensor({boundary_array.size(0)});
    TensorUtils::fill_range_cuda_(ba_row_i_to_bm_col_i); 
    ba_row_i_to_bm_col_i += n_simplices_by_dim.at(0); 

    //8. returning ... 
    ret.push_back(boundary_array); 
    ret.push_back(ba_row_i_to_bm_col_i);
    ret.push_back(simplex_dimension); 
    ret.push_back(sorted_filtration_values_vector);  

    return ret;
}


std::vector<std::vector<Tensor>> calculate_persistence_output_to_barcode_tensors(
    const std::vector<std::vector<Tensor>>& calculate_persistence_output,
    const Tensor & filtration_values){
    std::vector<std::vector<Tensor>> ret; 

    std::vector<Tensor> non_essential_barcodes; 
    {
        auto non_essentials = calculate_persistence_output.at(0);
        Tensor birth_death_i, births, birth_i, deaths, death_i, barcodes, i_birth_ne_death; 
        for (int i = 0; i < non_essentials.size(); i++){

            birth_death_i = non_essentials.at(i); 

            if(birth_death_i.numel() == 0){
                barcodes = filtration_values.type().tensor({0, 2}); 
            }
            else {
                birth_i = birth_death_i.slice(1, 0, 1).squeeze(); 
                births = filtration_values.index_select(0, birth_i);

                death_i = birth_death_i.slice(1, 1, 2).squeeze();
                deaths = filtration_values.index_select(0, death_i);

                i_birth_ne_death = births.ne(deaths).nonzero().squeeze(); 
                births = births.index_select(0, i_birth_ne_death);
                deaths = deaths.index_select(0, i_birth_ne_death);

                barcodes = stack({births, deaths}, 1); 

                
            }
            non_essential_barcodes.push_back(barcodes); 
        }
        ret.push_back(non_essential_barcodes);    
    }

    std::vector<Tensor> essential_barcodes; 
    {   
        auto essentials = calculate_persistence_output.at(1); 
        Tensor birth_i, births, barcodes; 
        for (int i = 0; i < essentials.size(); i++){

            birth_i = essentials.at(i).squeeze(); 

            if (birth_i.numel() == 0){
                barcodes = filtration_values.type().tensor({0, 1});
            }
            else {
                barcodes = filtration_values.index_select(0, birth_i); 
            }
            
            essential_barcodes.push_back(barcodes); 
        }
        ret.push_back(essential_barcodes); 
    }

    return ret; 
}


std::vector<std::vector<Tensor>> vr_l1_persistence(
    const Tensor& point_cloud,
    int64_t max_dimension, 
    double max_ball_radius){

    

    auto tmp = vr_l1_generate_calculate_persistence_args(
        point_cloud, max_dimension, max_ball_radius
    );

    auto pers = CalcPersCuda::calculate_persistence(
        tmp.at(0), tmp.at(1), tmp.at(2), max_dimension, -1
    );

    auto filtration_values = tmp.at(3); 
    auto ret = calculate_persistence_output_to_barcode_tensors(pers, filtration_values); 

    return ret;
}


} // namespace VRCompCuda 
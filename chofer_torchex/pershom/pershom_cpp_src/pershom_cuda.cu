#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <limits>


using namespace at;


#pragma region find_merge_pairings


namespace {

template <typename scalar_t>
__global__ void find_left_slicings_indices_cuda_kernel(
  scalar_t* __restrict__ input,
  scalar_t* __restrict__ output, 
  size_t input_size){ 

    const int index_middle = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_left   = index_middle - 1;
    const int index_right  = index_middle + 1;
    
    if (index_right < input_size){

      const int value_left = (index_left != -1) ? input[index_left] : -1; //OPTIMIZE: if we could
      // pad input with -1 on the left this conditional would be obsolete
      const int value_middle = input[index_middle];
      const int value_right = input[index_right];
      if (value_left != value_middle
          && 
          value_middle == value_right){
        output[index_middle] = index_middle;
      }
    }    
  }


template <typename scalar_t>
__global__ void find_right_slicings_indices_cuda_kernel(
  scalar_t* __restrict__ input,
  scalar_t* __restrict__ output, 
  size_t input_size){ 

    const int index_left   = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_middle = index_left + 1;
    const int index_right  = index_middle + 1;
    
    if (index_middle < input_size){

      const int value_left = input[index_left];
      const int value_middle = input[index_middle];
      const int value_right = (index_right < input_size) ? input[index_right] : (value_middle+1);//OPTIMIZE: if we could
      // pad input with inf on the right this conditional would be obsolete
      if (value_left == value_middle
          && 
          value_middle != value_right){
        output[index_middle] = index_middle + 1;
      }
    }    
  }

} // namespace


/**
 * @brief Finds the indices for slicing the sorted pivots values. 
 * Example:
 *    pivots.sort(0)[0] = [-1, -1, 2, 2, 2, 4, 4] -> [[2, 4], [5, 6]]
 * 
 * @tparam scalar_t 
 * @param pivots 
 * @return Tensor 
 */
template <typename scalar_t>
Tensor find_slicing_indices_cuda_kernel_call(
    Tensor pivots) {
  Tensor output = zeros_like(pivots).fill_(-1);
  const int threads_per_block = 256;
  const int blocks = pivots.size(0)/threads_per_block + 1;

  find_left_slicings_indices_cuda_kernel<scalar_t><<<blocks, threads_per_block>>>(
    pivots.data<scalar_t>(), 
    output.data<scalar_t>(),
    pivots.size(0));

  find_right_slicings_indices_cuda_kernel<scalar_t><<<blocks, threads_per_block>>>(
    pivots.data<scalar_t>(), 
    output.data<scalar_t>(),
    pivots.size(0));

  output = output.masked_select(output.ge(0));
  output = output.view(IntList({output.size(0)/2, 2}));

  return output;
}  


__global__ void extract_slicings_cuda_kernel(
  int64_t* p_input,
  int32_t* p_slicings, 
  int64_t* p_return_value, 
  int64_t return_value_size_0, 
  int64_t return_value_size_1){
    
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < return_value_size_0){
      auto p_return_value_row = p_return_value + thread_id * return_value_size_1;
      const int slice_start = *(p_slicings + (thread_id * 2));
      const int slice_end = *(p_slicings + (thread_id * 2) + 1);

      for (int i = 0; i < slice_end - slice_start; i++){
        *(p_return_value_row + i) = *(p_input + slice_start + i);
      }
    }
}


__global__ void merge_pairings_from_extractd_sorted_slicings(
  int64_t* extracted_slices, 
  int64_t extracted_slices_size_0,
  int64_t extracted_slices_size_1,
  int32_t* lengths, 
  int64_t* row_offset_for_thread,
  int64_t* return_value){
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < extracted_slices_size_0){
      const int length = *(lengths + thread_id);
      const int row_offset = (thread_id > 0) ? *(row_offset_for_thread + thread_id - 1) : 0;
      auto extracted_slices_row = extracted_slices + thread_id*extracted_slices_size_1; 
      auto const first_col_value = *extracted_slices_row;

      auto return_value_row = return_value + 2*row_offset;
      for (int i = 0; i < length - 1; i++){
        *(return_value_row) = first_col_value;
        *(return_value_row + 1) = *(extracted_slices_row + i + 1);
        return_value_row = return_value_row + 2;
      }
    }
  }



Tensor merge_pairings_from_sort_ind_slicings(Tensor input, Tensor slicings){
  // ASSERTION input.dtype() == int64
  // ASSERTION slicings.dtype() == int32
  // ASSERTION all(input.ge(0))

  auto lengths = (slicings.slice(1, 1, 2) - slicings.slice(1, 0, 1)).contiguous();
  auto max_lengths = Scalar(lengths.max()).to<int>(); 
  Tensor extracted_slicings = input.type().empty({slicings.size(0), max_lengths});
  extracted_slicings.fill_(std::numeric_limits<int64_t>::max());

  const int threads_per_block_apply_slicings = 256;
  const int blocks_apply_slicings = slicings.size(0)/threads_per_block_apply_slicings + 1;
  extract_slicings_cuda_kernel<<<threads_per_block_apply_slicings, blocks_apply_slicings>>>(
    input.data<int64_t>(), 
    slicings.data<int32_t>(),
    extracted_slicings.data<int64_t>(),
    extracted_slicings.size(0),
    extracted_slicings.size(1)
  );

  auto extracted_slicings_sorted = std::get<0>(extracted_slicings.sort(1)).contiguous();
  // std::cout << extracted_slicings_sorted.slice(0, 0, 10).slice(1, 0, 10) << std::endl;

  auto lengths_minus_1 = lengths - lengths.type().scalarTensor(1);  
  auto row_offset_for_thread = lengths_minus_1.cumsum(0);
  // std::cout << row_offset_for_thread.slice(0, 0, 10).slice(1, 0, 10) << std::endl;

  auto merge_pairings_size_0 = Scalar(row_offset_for_thread[-1][0]).to<int>();
  auto merge_pairings = input.type().empty({merge_pairings_size_0, 2});
  merge_pairings.fill_(-1);

  const int threads_per_block = 256;
  const int blocks = extracted_slicings_sorted.size(0)/threads_per_block + 1;

  merge_pairings_from_extractd_sorted_slicings<<<threads_per_block, blocks>>>(
      extracted_slicings_sorted.data<int64_t>(), 
      extracted_slicings_sorted.size(0),
      extracted_slicings_sorted.size(1), 
      lengths.data<int32_t>(), 
      row_offset_for_thread.data<int64_t>(),
      merge_pairings.data<int64_t>()
  );

  return merge_pairings;
  
}

class NoPairsException{
public:
  NoPairsException() {}
 ~NoPairsException() {}
};


Tensor find_merge_pairings_cuda(
  Tensor pivots,
  int max_pairs = -1 ){

    if (max_pairs < 1){
      max_pairs = std::numeric_limits<int>::max();
    }
    auto sort_res = pivots.sort(0);
    auto sort_val = std::get<0>(sort_res);
    auto sort_ind = std::get<1>(sort_res);

    // remove columns with undefined pivot (i.e. -1)
    auto mask = sort_val.ge(0);
    sort_val = sort_val.masked_select(mask);
    sort_ind = sort_ind.masked_select(mask);

    auto slicings = find_slicing_indices_cuda_kernel_call<int32_t>(sort_val).contiguous();

    Tensor merge_pairs; 
    if (slicings.size(0) != 0){         

      merge_pairs = merge_pairings_from_sort_ind_slicings(sort_ind, slicings);
      // We sort the pairs such that pairs with smaller index come first.
      // This improves performance???
      if (merge_pairs.size(0) > max_pairs){

        sort_res = merge_pairs.slice(1, 0, 1).sort(0);
        sort_ind = std::get<1>(sort_res);
        sort_ind = sort_ind.slice(0, 0, max_pairs).squeeze();

        merge_pairs = merge_pairs.index_select(0, sort_ind);
        merge_pairs = merge_pairs.contiguous();
      }
    }
    else{
      throw NoPairsException();
    }

   return merge_pairs;
}


#pragma endregion 


#pragma region merge_columns


namespace{


  template <typename scalar_t>
  __device__ void merge_one_column_s(
    scalar_t* p_merger, 
    scalar_t* p_target, // the position of the target column, set to -1
    scalar_t* p_target_cache, // contains the copied values of target column 
    int boundary_array_size_1, 
    int* d_boundary_array_needs_resize 
  ){    
    // Assertion: descending_sorted_boundary_array[:, -1] == -1 

    int p_target_increment_count = 0;

    while (true){
      if (*p_merger == -1 && *p_target_cache == -1){
        // both are -1, we have reached the end of meaningful entries -> break
        break;
      }

      if (*p_merger == *p_target_cache){
        // both values are the same but not -1 -> we eliminate 
        p_target_cache++;
        p_merger++;
      }
      else {

        if (*p_merger > *p_target_cache){
          //merger value is greater -> we take it 
          *p_target = *p_merger;
          p_merger++;
        }
        else
        {
          //target value is greate -> we take it 
          *p_target = *p_target_cache;
          p_target_cache++;
        }

        p_target++;  
        p_target_increment_count += 1;
      }          
    }

    if (p_target_increment_count > boundary_array_size_1/2){
      *d_boundary_array_needs_resize = 1; 
    }
  }


  template <typename scalar_t>
  __global__ void merge_columns_cuda_kernel(
      scalar_t* descending_sorted_boundary_array,
      size_t descending_sorted_boundary_array_size_1, 
      scalar_t* cache, 
      int64_t* merge_pairings,
      size_t merge_pairings_size_0, 
      int* d_boundary_array_needs_resize
  ){
    //ASSERTION: cache.size(1) == descending_sorted_boundary_array.size(1)
    const int thread_id = blockIdx.x*blockDim.x + threadIdx.x;   

    if (thread_id < merge_pairings_size_0){  

      const int filt_id_merger = merge_pairings[thread_id * 2];
      const int filt_id_target = merge_pairings[thread_id * 2 + 1];

      merge_one_column_s<int32_t>(
        descending_sorted_boundary_array + (filt_id_merger * descending_sorted_boundary_array_size_1),
        descending_sorted_boundary_array + (filt_id_target * descending_sorted_boundary_array_size_1),
        cache + (thread_id * descending_sorted_boundary_array_size_1), 
        descending_sorted_boundary_array_size_1,
        d_boundary_array_needs_resize
      );
    }
  }
  

} //namespace


template <typename scalar_t>
void merge_columns_cuda_kernel_call(
  Tensor descending_sorted_boundary_array,
  Tensor merge_pairings, 
  int* h_boundary_array_needs_resize
)
{
  const int threads_per_block = 32;
  const int blocks = merge_pairings.size(0)/threads_per_block + 1;

  auto targets = merge_pairings.slice(1, 1).squeeze();  
  
  // fill cache for merging ... 
  //  TODO optimize: we do not need all columns it is enough to take des...array.size(1)/2 + 1 
  //  ATTENTION if we do this we have to inform merge_columns_cuda_kernel about this!!!
  auto cache = descending_sorted_boundary_array.index_select(0, targets);
  
  auto size = sizeof(int);
  int* d_boundary_array_needs_resize;
  hipMalloc(&d_boundary_array_needs_resize, size);
  hipMemcpy(d_boundary_array_needs_resize, h_boundary_array_needs_resize, size, hipMemcpyHostToDevice);

  // reset content of target columns 
  descending_sorted_boundary_array.index_fill_(0, targets, -1);

  merge_columns_cuda_kernel<int32_t><<<blocks, threads_per_block>>>(
    descending_sorted_boundary_array.data<int32_t>(), 
    descending_sorted_boundary_array.size(1), 
    cache.data<int32_t>(),
    merge_pairings.data<int64_t>(), 
    merge_pairings.size(0), 
    d_boundary_array_needs_resize
  );

  hipDeviceSynchronize();
  hipMemcpy(h_boundary_array_needs_resize, d_boundary_array_needs_resize, size, hipMemcpyDeviceToHost);

  hipFree(d_boundary_array_needs_resize);
}


Tensor resize_boundary_array(
  Tensor descending_sorted_boundary_array){
    auto tmp = empty_like(descending_sorted_boundary_array);
    tmp.fill_(-1);
    auto new_ba = cat(TensorList({descending_sorted_boundary_array, tmp}), 1);
    return new_ba.contiguous();
}


Tensor merge_columns_cuda(
  Tensor descending_sorted_boundary_array, 
  Tensor merge_pairings){   
   
    int boundary_array_needs_resize = 0;
    int* h_boundary_array_needs_resize = &boundary_array_needs_resize;    

    merge_columns_cuda_kernel_call<int32_t>(
      descending_sorted_boundary_array,
      merge_pairings, 
      h_boundary_array_needs_resize
    );
  
    if (*h_boundary_array_needs_resize == 1){
      descending_sorted_boundary_array = resize_boundary_array(descending_sorted_boundary_array);
    }
    
    return descending_sorted_boundary_array;
  }


#pragma endregion


#pragma region read_barcodes


namespace {
  template<typename scalar_t>
  __global__ void fill_range_kernel(scalar_t* out, int64_t out_numel){
    auto index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < out_numel){
      out[index] = index;
    }
  }
}

void fill_range_cuda_(Tensor t){
  const int threads_per_block = 256;
  const int blocks = t.numel()/threads_per_block + 1;

  fill_range_kernel<int32_t><<<blocks, threads_per_block>>>(t.data<int32_t>(), t.numel());
}

std::vector<std::vector<Tensor> > read_barcodes_cuda(
  Tensor pivots, 
  Tensor simplex_dimension, 
  int max_dimension){
    std::vector<Tensor> ret_non_ess; 
    std::vector<Tensor> ret_ess;
    simplex_dimension = simplex_dimension.unsqueeze(1);    

    auto range = empty_like(pivots);
    fill_range_cuda_(range); 

    auto pool_for_barcodes_non_essential = cat({pivots, range}, 1);
    auto mask_pivot = pivots.ge(0);
    
    // all dimenions mask non essential ... 
    auto mask_non_essential = mask_pivot.expand({-1, 2});

    // all dimensions mask essential ...
    auto mask_no_pivot = pivots.le(-1); 

    auto mask_rows_with_no_lowest_one = ones_like(mask_no_pivot);
    auto row_indices_with_lowest_one = pivots.masked_select(mask_pivot).toType(ScalarType::Long);
    mask_rows_with_no_lowest_one.index_fill_(0, row_indices_with_lowest_one, 0);

    auto mask_ess = mask_no_pivot.__and__(mask_rows_with_no_lowest_one);

    for (int dim=0; dim <= max_dimension; dim++){
      
      // non essentials ...
      auto mask_dim = simplex_dimension.eq(dim + 1);
      auto mask_non_essential_dim = mask_non_essential.__and__(mask_dim.expand({-1, 2}));
      auto barcodes_non_essential_dim = pool_for_barcodes_non_essential.masked_select(mask_non_essential_dim).view({-1, 2});
      
      ret_non_ess.push_back(barcodes_non_essential_dim);
      
      // essentials ...
      auto mask_dim_ess = simplex_dimension.eq(dim);
      auto mask_essential_dim = mask_ess.__and__(mask_dim_ess); 
      auto barcode_birth_times_essential_dim = range.masked_select(mask_essential_dim).view({-1, 1});

      ret_ess.push_back(barcode_birth_times_essential_dim);
    } 

    return std::vector<std::vector<Tensor> >({ret_non_ess, ret_ess});
  }


#pragma endregion 


std::vector<std::vector<Tensor> > calculate_persistence_cuda(  
  Tensor descending_sorted_boundary_array, 
  Tensor simplex_dimension,
  int max_dimension,
  int max_pairs = -1
  ) {

  int iterations = 0;
  Tensor pivots, merge_pairings;

  while(true){
    pivots = descending_sorted_boundary_array.slice(1, 0, 1).contiguous();

    try{

      merge_pairings = find_merge_pairings_cuda(pivots, max_pairs);   

    }
    catch(NoPairsException& e){

      std::cout << "Reached end of reduction after " << iterations << " iterations" << std::endl;
      break;

    }
    
    descending_sorted_boundary_array = merge_columns_cuda(descending_sorted_boundary_array, merge_pairings);
    iterations++;

  }

  auto barcodes = read_barcodes_cuda(pivots, simplex_dimension, max_dimension);
  return barcodes;
}

//-----------devel

// namespace {
//   __global__ void my_test_kernel(Tensor t){
//     int x = 0;
//     auto index = blockIdx.x * blockDim.x + threadIdx.x; 
//     t[index][2] = 1;
//   }
// }

Tensor my_test_f_cuda(Tensor t){
  auto ret = zeros_like(t);

  // my_test_kernel<<<1, 32>>>(t);

  return ret;
}

//-----------
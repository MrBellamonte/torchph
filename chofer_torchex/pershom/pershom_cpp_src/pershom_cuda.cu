#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>


using namespace at;


#pragma region find_merge_pairings


namespace {

template <typename scalar_t>
__global__ void find_left_plateau_indices_cuda_kernel(
  scalar_t* __restrict__ input,
  scalar_t* __restrict__ output, 
  size_t input_size){ 

    const int index_middle = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_left   = index_middle - 1;
    const int index_right  = index_middle + 1;
    
    if (index_right < input_size){

      const int value_left = (index_left != -1) ? input[index_left] : -1; //OPTIMIZE: if we could
      // pad input with -1 on the left this conditional would be obsolete
      const int value_middle = input[index_middle];
      const int value_right = input[index_right];
      if (value_left != value_middle
          && 
          value_middle == value_right){
        output[index_middle] = index_middle;
      }
    }    
  }


template <typename scalar_t>
__global__ void find_right_plateau_indices_cuda_kernel(
  scalar_t* __restrict__ input,
  scalar_t* __restrict__ output, 
  size_t input_size){ 

    const int index_left   = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_middle = index_left + 1;
    const int index_right  = index_middle + 1;
    
    if (index_middle < input_size){

      const int value_left = input[index_left];
      const int value_middle = input[index_middle];
      const int value_right = (index_right < input_size) ? input[index_right] : (value_middle+1);//OPTIMIZE: if we could
      // pad input with inf on the right this conditional would be obsolete
      if (value_left == value_middle
          && 
          value_middle != value_right){
        output[index_middle] = index_middle;
      }
    }    
  }


} // namespace


class NoPairsException{
public:
  NoPairsException() {}
 ~NoPairsException() {}
};


template <typename scalar_t>
Tensor find_slicing_indices_cuda_kernel_call(
    Tensor input) {
  Tensor output = zeros_like(input).fill_(-1);
  
  const int threads_per_block = 256;
  const int blocks = input.size(0)/threads_per_block + 1;

  find_left_plateau_indices_cuda_kernel<scalar_t><<<blocks, threads_per_block>>>(
    input.data<scalar_t>(), 
    output.data<scalar_t>(),
    input.size(0));

  find_right_plateau_indices_cuda_kernel<scalar_t><<<blocks, threads_per_block>>>(
    input.data<scalar_t>(), 
    output.data<scalar_t>(),
    input.size(0));

  output = output.masked_select(output.ge(0));
  output = output.view(IntList({output.size(0)/2, 2}));

  return output;
}     


Tensor find_merge_pairings_cuda(
  Tensor pivots,
  int max_pairs){

    // std::cout << pivots << std::endl;
    auto sort_res = pivots.sort(0);
    auto sort_val = std::get<0>(sort_res);
    auto sort_ind = std::get<1>(sort_res);

    // remove columns with undefined pivot (i.e. -1)
    auto mask = sort_val.ge(0);
    sort_val = sort_val.masked_select(mask);
    sort_ind = sort_ind.masked_select(mask);

    // std::vector<Tensor> l({sort_val, sort_ind.type_as(sort_val)});
    // std::cout << stack(l, 1) << std::endl;

    auto slicings = find_slicing_indices_cuda_kernel_call<int32_t>(sort_val);
    // std::cout << slicings << std::endl;

    int pairing_counter = 0;
    std::vector<Tensor> pairing_tensors; 
    for (int i=0; i<slicings.size(0); i++){

      if (pairing_counter > max_pairs){
        break;
      }

      auto slicing_i = slicings[i];
      auto begin = Scalar(slicing_i[0]).to<int>(); //OPTIMIZE: can this conversion be improved?
      auto end = Scalar(slicing_i[1]).to<int>() + 1;
      auto slice = sort_ind.slice(0, begin, end);
      slice = std::get<0>(slice.sort(0));

      auto col_2 = slice.slice(0, 1);
      auto col_1 = slice[0].expand_as(col_2);
      auto pairing_tensor = stack(std::vector<Tensor>({col_1, col_2}), 1);

      pairing_counter += pairing_tensor.size(0);

      pairing_tensors.push_back(pairing_tensor);
    }

    Tensor merge_pairs;
    if (pairing_tensors.size() != 0){      
      merge_pairs = cat(pairing_tensors, 0);     

      // We sort the pairs such that pairs with smaller index come first.
      // This improves performance???
      if (merge_pairs.size(0) > max_pairs){

        sort_res = merge_pairs.slice(1, 0, 1).sort(0);
        sort_ind = std::get<1>(sort_res);
        sort_ind = sort_ind.slice(0, 0, max_pairs).squeeze();

        merge_pairs = merge_pairs.index_select(0, sort_ind);
        merge_pairs = merge_pairs.contiguous();
      }
    }
    else{
      throw NoPairsException();
    }

   return merge_pairs;
}


#pragma endregion 


#pragma region merge_columns


namespace{


  template <typename scalar_t>
  __device__ void merge_one_column_s(
    scalar_t* p_merger, 
    scalar_t* p_target, // the position of the target column, set to -1
    scalar_t* p_target_cache, // contains the copied values of target column 
    int boundary_array_size_1, 
    int* d_boundary_array_needs_resize 
  ){    
    // Assertion: descending_sorted_boundary_array[:, -1] == -1 

    int p_target_increment_count = 0;

    while (true){
      if (*p_merger == -1 && *p_target_cache == -1){
        // both are -1, we have reached the end of meaningful entries -> break
        break;
      }

      if (*p_merger == *p_target_cache){
        // both values are the same but not -1 -> we eliminate 
        p_target_cache++;
        p_merger++;
      }
      else {

        if (*p_merger > *p_target_cache){
          //merger value is greater -> we take it 
          *p_target = *p_merger;
          p_merger++;
        }
        else
        {
          //target value is greate -> we take it 
          *p_target = *p_target_cache;
          p_target_cache++;
        }

        p_target++;  
        p_target_increment_count += 1;
      }          
    }

    if (p_target_increment_count > boundary_array_size_1/2){
      *d_boundary_array_needs_resize = 1; 
    }
  }


  template <typename scalar_t>
  __global__ void merge_columns_cuda_kernel(
      scalar_t* descending_sorted_boundary_array,
      size_t descending_sorted_boundary_array_size_1, 
      scalar_t* cache, 
      int64_t* merge_pairings,
      size_t merge_pairings_size_0, 
      int* d_boundary_array_needs_resize
  ){
    //ASSERTION: cache.size(1) == descending_sorted_boundary_array.size(1)
    const int thread_id = blockIdx.x*blockDim.x + threadIdx.x;   

    if (thread_id < merge_pairings_size_0){  

      const int filt_id_merger = merge_pairings[thread_id * 2];
      const int filt_id_target = merge_pairings[thread_id * 2 + 1];

      merge_one_column_s<int32_t>(
        descending_sorted_boundary_array + (filt_id_merger * descending_sorted_boundary_array_size_1),
        descending_sorted_boundary_array + (filt_id_target * descending_sorted_boundary_array_size_1),
        cache + (thread_id * descending_sorted_boundary_array_size_1), 
        descending_sorted_boundary_array_size_1,
        d_boundary_array_needs_resize
      );
    }
  }
  

} //namespace


template <typename scalar_t>
void merge_columns_cuda_kernel_call(
  Tensor descending_sorted_boundary_array,
  Tensor merge_pairings, 
  int* h_boundary_array_needs_resize
)
{
  const int threads_per_block = 32;
  const int blocks = merge_pairings.size(0)/threads_per_block + 1;

  auto targets = merge_pairings.slice(1, 1).squeeze();  
  
  // fill cache for merging ... 
  //  TODO optimize: we do not need all columns it is enough to take des...array.size(1)/2 + 1 
  //  ATTENTION if we do this we have to inform merge_columns_cuda_kernel about this!!!
  auto cache = descending_sorted_boundary_array.index_select(0, targets);
  
  auto size = sizeof(int);
  int* d_boundary_array_needs_resize;
  hipMalloc(&d_boundary_array_needs_resize, size);
  hipMemcpy(d_boundary_array_needs_resize, h_boundary_array_needs_resize, size, hipMemcpyHostToDevice);

  // reset content of target columns 
  descending_sorted_boundary_array.index_fill_(0, targets, -1);

  merge_columns_cuda_kernel<int32_t><<<blocks, threads_per_block>>>(
    descending_sorted_boundary_array.data<int32_t>(), 
    descending_sorted_boundary_array.size(1), 
    cache.data<int32_t>(),
    merge_pairings.data<int64_t>(), 
    merge_pairings.size(0), 
    d_boundary_array_needs_resize
  );

  hipDeviceSynchronize();
  hipMemcpy(h_boundary_array_needs_resize, d_boundary_array_needs_resize, size, hipMemcpyDeviceToHost);

  hipFree(d_boundary_array_needs_resize);
}


Tensor resize_boundary_array(
  Tensor descending_sorted_boundary_array){
    auto tmp = empty_like(descending_sorted_boundary_array);
    tmp.fill_(-1);
    auto new_ba = cat(TensorList({descending_sorted_boundary_array, tmp}), 1);
    return new_ba.contiguous();
}


Tensor merge_columns_cuda(
  Tensor descending_sorted_boundary_array, 
  Tensor merge_pairings){   
   
    int boundary_array_needs_resize = 0;
    int* h_boundary_array_needs_resize = &boundary_array_needs_resize;    

    merge_columns_cuda_kernel_call<int32_t>(
      descending_sorted_boundary_array,
      merge_pairings, 
      h_boundary_array_needs_resize
    );
  
    if (*h_boundary_array_needs_resize == 1){
      descending_sorted_boundary_array = resize_boundary_array(descending_sorted_boundary_array);
    }
    
    return descending_sorted_boundary_array;
  }


#pragma endregion


#pragma region read_barcodes


namespace {
  template<typename scalar_t>
  __global__ void fill_range_kernel(scalar_t* out, int64_t out_numel){
    auto index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < out_numel){
      out[index] = index;
    }
  }
}

void fill_range_cuda_(Tensor t){
  const int threads_per_block = 256;
  const int blocks = t.numel()/threads_per_block + 1;

  fill_range_kernel<int32_t><<<blocks, threads_per_block>>>(t.data<int32_t>(), t.numel());
}

std::vector<std::vector<Tensor> > read_barcodes_cuda(
  Tensor pivots, 
  Tensor column_dimension, 
  int max_dimension){
    std::vector<Tensor> ret_non_ess; 
    std::vector<Tensor> ret_ess;
    column_dimension = column_dimension.unsqueeze(1);    

    auto range = empty_like(pivots);
    fill_range_cuda_(range); 

    auto pool_for_barcodes_non_essential = cat({pivots, range}, 1);
    auto mask_pivot = pivots.ge(0);
    
    // all dimenions mask non essential ... 
    auto mask_non_essential = mask_pivot.expand({-1, 2});

    // all dimensions mask essential ...
    auto mask_no_pivot = pivots.le(-1); 

    auto mask_rows_with_no_lowest_one = ones_like(mask_no_pivot);
    auto row_indices_with_lowest_one = pivots.masked_select(mask_pivot).toType(ScalarType::Long);
    mask_rows_with_no_lowest_one.index_fill_(0, row_indices_with_lowest_one, 0);

    auto mask_ess = mask_no_pivot.__and__(mask_rows_with_no_lowest_one);

    for (int dim=0; dim <= max_dimension; dim++){
      
      // non essentials ...
      auto mask_dim = column_dimension.eq(dim + 1);
      auto mask_non_essential_dim = mask_non_essential.__and__(mask_dim.expand({-1, 2}));
      auto barcodes_non_essential_dim = pool_for_barcodes_non_essential.masked_select(mask_non_essential_dim).view({-1, 2});
      
      ret_non_ess.push_back(barcodes_non_essential_dim);
      
      // essentials ...
      auto mask_dim_ess = column_dimension.eq(dim);
      auto mask_essential_dim = mask_ess.__and__(mask_dim_ess); 
      auto barcode_birth_times_essential_dim = range.masked_select(mask_essential_dim).view({-1, 1});

      ret_ess.push_back(barcode_birth_times_essential_dim);
    } 

    return std::vector<std::vector<Tensor> >({ret_non_ess, ret_ess});
  }


#pragma endregion 


std::vector<std::vector<Tensor> > calculate_persistence_cuda(  
  Tensor descending_sorted_boundary_array, 
  Tensor column_dimension,
  int max_dimension,
  int max_pairs
  ) {

  int iterations = 0;
  Tensor pivots, merge_pairings;

  while(true){
    pivots = descending_sorted_boundary_array.slice(1, 0, 1).contiguous();

    try{

      merge_pairings = find_merge_pairings_cuda(pivots, max_pairs);   

    }
    catch(NoPairsException& e){

      std::cout << "Reached end of reduction after " << iterations << " iterations" << std::endl;
      break;

    }
    
    descending_sorted_boundary_array = merge_columns_cuda(descending_sorted_boundary_array, merge_pairings);
    iterations++;

  }

  auto barcodes = read_barcodes_cuda(pivots, column_dimension, max_dimension);
  return barcodes;
}

//-----------devel

// namespace {
//   __global__ void my_test_kernel(Tensor t){
//     int x = 0;
//     auto index = blockIdx.x * blockDim.x + threadIdx.x; 
//     t[index][2] = 1;
//   }
// }

Tensor my_test_f_cuda(Tensor t){
  auto ret = zeros_like(t);

  // my_test_kernel<<<1, 32>>>(t);

  return ret;
}

//-----------